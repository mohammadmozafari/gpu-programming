#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

// GPU kernel
__global__ void add(float* a, float* b, float* c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

// CPU implementation
void add_cpu(float* a, float* b, float* c, int N) {
    for (int i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int N = 1 << 24;                        // ~16 million elements
    size_t size = N * sizeof(float);

    float *a = (float*)malloc(size);
    float *b = (float*)malloc(size);
    float *c_gpu = (float*)malloc(size);
    float *c_cpu = (float*)malloc(size);
    float gpu_time = 0.0f;

    // Seed RNG
    srand((unsigned int)time(NULL));

    // Random initialization
    for (int i = 0; i < N; i++) {
        a[i] = (float)(rand() % 1000) / 100.0f;  // values in [0,10)
        b[i] = (float)(rand() % 1000) / 100.0f;
    }

    // --- CPU timing
    clock_t start_cpu = clock();
    add_cpu(a, b, c_cpu, N);
    clock_t end_cpu = clock();
    double cpu_time = (double)(end_cpu - start_cpu) / CLOCKS_PER_SEC;

    // --- GPU memory allocation
    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // --- GPU timing (CUDA events)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threads_per_block = 256;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
    hipEventRecord(start);
    add<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    
    hipMemcpy(c_gpu, d_c, size, hipMemcpyDeviceToHost);
    
    // --- Check correctness
    for (int i = 0; i < N; i++) {
        if (fabs(c_cpu[i] - c_gpu[i]) > 1e-6) {
            printf("Mismatch at index %d!\n", i);
        }
    }

    // --- Print timing
    printf("\nCPU time: %f ms\n", cpu_time * 1000);
    printf("GPU time: %f ms\n", gpu_time);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c_cpu);
    free(c_gpu);

    return 0;
}
