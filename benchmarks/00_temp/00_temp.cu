#include <stdio.h>
#include <hip/hip_runtime.h>

void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__ void hello_cuda() {
    printf("Hello from GPU!\n");
}

int main() {
    hello_cuda<<<1, 1>>>();
    // Check for errors after the kernel launch
    checkCudaErrors(hipGetLastError());
    
    // Wait for the kernel to finish and check for any runtime errors
    checkCudaErrors(hipDeviceSynchronize());
    fflush(stdout); // Add this line to flush the output buffer
    return 0;
}